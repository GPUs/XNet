#include "hip/hip_runtime.h"
/*
 * set_value.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "set_value.cuh"

namespace utils {

__global__ void setValue(float* x, int n, float val) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
		x[i] = val;
}

void setGpuValue(float* x, int n, float val) {
	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	setValue<<<blocksPerGrid, threadsPerBlock>>>(x, n, val);
}

void setGpuNormalValue(float* x, int n, float mean, float stddev) {
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));
	hiprandGenerateNormal(generator, x, n, mean, stddev);
	hiprandDestroyGenerator(generator);
}

void setGpuNormalValue(float* x, int n) {
	setGpuNormalValue(x, n, 0, sqrt(3.0f / n));
}

}
