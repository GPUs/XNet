#include "hip/hip_runtime.h"
/*
 * set_value.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "set_value.cuh"

using namespace global;

namespace utils {

__global__ void setValue(float* x, int n, float val) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
		x[i] = val;
}

__global__ void dropout(float* x, float* t, int n, float threshold) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
		if (t[i] < threshold)
			x[i] = 0;
}

__global__ void scale(float* x, int n, float epsilon) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
		x[i] = x[i] * epsilon * 2 - epsilon;
}

void setGpuValue(float* x, int n, float val) {
	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	setValue<<<blocksPerGrid, threadsPerBlock>>>(x, n, val);
}

void setGpuUniformValue(float* x, int n, int input_size, int output_size) {
	float epsilon = std::sqrt(6.0 / (input_size + output_size));
	setGpuUniformValue(x, n, epsilon);
}

void setGpuUniformValue(float* x, int n, float epsilon) {
	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));
	hiprandGenerateUniform(generator, x, n);
	scale<<<blocksPerGrid, threadsPerBlock>>>(x, n, epsilon);
	hiprandDestroyGenerator(generator);
}

void setGpuNormalValue(float* x, int n, float mean, float stddev) {
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));
	hiprandGenerateNormal(generator, x, n, mean, stddev);
	hiprandDestroyGenerator(generator);
}

void setGpuNormalValue(float* x, int n, int scale) {
	if (scale == -1)
		scale = n;
	setGpuNormalValue(x, n, 0, sqrt(0.3f / scale));
}

void dropGpuValue(float *x, int n, float dropout_rate) {
	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));
	float* t;
	hipMalloc((void**)&t, sizeof(float) * n);
	hiprandGenerateUniform(generator, t, n);
	dropout<<<blocksPerGrid, threadsPerBlock>>>(x, t, n, dropout_rate);
	hipFree(t);
	hiprandDestroyGenerator(generator);
}

void scaleGpuValue(float *x, int n, float scale) {
	hipblasSscal(cublasHandle, n, &scale, x, 1);
}

}
