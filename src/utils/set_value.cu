#include "hip/hip_runtime.h"
/*
 * set_value.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "set_value.cuh"

namespace utils {

void setGpuNormalValue(float* x, int n, float mean, float stddev) {
	hiprandGenerator_t generator;
	callCurand(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
	callCurand(hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL)));
	callCurand(hiprandGenerateNormal(generator, x, n, mean, stddev));
	callCurand(hiprandDestroyGenerator(generator));
}

void setGpuNormalValue(float* x, int n) {
	setGpuNormalValue(x, n, 0, sqrt(3.0 / n));
}

}
