/*
 * neuron.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "neuron.cuh"

using namespace global;

namespace layer {

Neuron::Neuron(Layer* _prev, int _output_size) {
	prev = _prev;
	prev->next = this;

	int _n, _c, _h, _w, _tmp;
	hipdnnDataType_t _t;
	callCudnn(hipdnnGetTensor4dDescriptor(prev->t_data, &_t, &_n, &_c, &_h, &_w, &_tmp,
			&_tmp, &_tmp, &_tmp));
	batch = _n;
	input_size = _c;
	output_size = _output_size;
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			_n, output_size, 1, 1));
	callCuda(hipMalloc(&data, sizeof(float) * _n * output_size));
	callCuda(hipMalloc(&diff, sizeof(float) * _n * output_size));

	callCuda(hipMalloc(&param, sizeof(float) * _c * output_size));
	callCuda(hipMalloc(&param_bias, sizeof(float) * output_size));
	callCuda(hipMalloc(&gradient, sizeof(float) * _c * output_size));

	callCuda(hipMalloc(&one, sizeof(float) * batch));

}

Neuron::~Neuron() {
	callCudnn(hipdnnDestroyTensorDescriptor(t_data));
	callCuda(hipFree(data));
	callCuda(hipFree(diff));
	callCuda(hipFree(param));
	callCuda(hipFree(param_bias));
	callCuda(hipFree(gradient));
	callCuda(hipFree(one));
}

void Neuron::forward() {
	float a = 1;
	float b = 0;
	callCuda(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, output_size, batch,
			input_size,	&a, param, input_size, prev->data, input_size, &b, data,
			output_size));
	callCuda(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, output_size, batch,
			1, &a, param_bias, output_size,	one, 1,	&a,	data, output_size));
}

void Neuron::backward() {

}

void Neuron::update() {

}

}
