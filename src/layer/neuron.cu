/*
 * neuron.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "neuron.cuh"

using namespace global;

namespace layer {

Neuron::Neuron(Layer* _prev, int _output_size, float dropout_rate, float alpha,
		float sigma): Layer(alpha) {
	prev = _prev;
	prev->next = this;

	int _n, _c, _h, _w, _tmp;
	hipdnnDataType_t _t;
	callCudnn(hipdnnGetTensor4dDescriptor(prev->t_data, &_t, &_n, &_c, &_h, &_w, &_tmp,
			&_tmp, &_tmp, &_tmp));
	batch = _n;
	input_size = _c * _h * _w;
	output_size = _output_size;
	callCudnn(hipdnnCreateTensorDescriptor(&t_data));
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			batch, output_size, 1, 1));
	data_size = batch * output_size;
	callCuda(hipMalloc(&data, sizeof(float) * data_size));
	callCuda(hipMalloc(&tmp_data, sizeof(float) * data_size));
	callCuda(hipMalloc(&diff, sizeof(float) * prev->data_size));
	callCuda(hipMalloc(&tmp_diff, sizeof(float) * data_size));

	param_size = input_size * output_size;
	param_bias_size = output_size;
	callCuda(hipMalloc(&param, sizeof(float) * param_size));
	callCuda(hipMalloc(&param_bias, sizeof(float) * param_bias_size));
	callCuda(hipMalloc(&gradient, sizeof(float) * param_size));
	callCuda(hipMalloc(&gradient_bias, sizeof(float) * param_bias_size));

	utils::setGpuNormalValue(param, param_size, 0, sigma);
	utils::setGpuNormalValue(param_bias, param_bias_size, 0, sigma);

	callCuda(hipMalloc(&one, sizeof(float) * batch));
	utils::setGpuValue(one, batch, 1);

	this->dropout_rate = dropout_rate;
}

Neuron::~Neuron() {
	callCudnn(hipdnnDestroyTensorDescriptor(t_data));
	callCuda(hipFree(data));
	callCuda(hipFree(tmp_data));
	callCuda(hipFree(diff));
	callCuda(hipFree(tmp_diff));
	callCuda(hipFree(param));
	callCuda(hipFree(param_bias));
	callCuda(hipFree(gradient));
	callCuda(hipFree(gradient_bias));
	callCuda(hipFree(one));
}

void Neuron::forward(bool train) {
	float a = 1;
	float b = 0;
	dropout(train);
	callCuda(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, output_size, batch,
			input_size,	&a, param, input_size, prev->data, input_size, &b, tmp_data,
			output_size));
	callCuda(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, output_size, batch,
			1, &a, param_bias, output_size,	one, 1,	&a,	tmp_data, output_size));
	forward_activation();
}

void Neuron::backward() {
	float a = 1;
	float b = 0;
	backward_activation();
	callCuda(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, input_size,
			output_size, batch, &a, prev->data, input_size, tmp_diff, output_size,
			&b, gradient, input_size));
	callCuda(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, output_size, batch,
			&a, tmp_diff, output_size, one, 1, &b, gradient_bias, 1));
	callCuda(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, input_size,
			batch, output_size, &a, param, input_size, tmp_diff, output_size,
			&b, diff, input_size));
}

void Neuron::update() {
	//utils::printGpuMatrix(prev->data, 10, 1, 10, 8);
	//utils::printGpuMatrix(param, 10, 1, 10, 9);
	//utils::printGpuMatrix(gradient, 10, 1, 10, 10);
	callCuda(hipblasSaxpy(cublasHandle, param_size, &alpha, gradient, 1, param, 1));
	callCuda(hipblasSaxpy(cublasHandle, param_bias_size,	&alpha,
			gradient_bias, 1, param_bias, 1));
}

void Neuron::dropout(bool train) {
	if (train)
		utils::dropGpuValue(prev->data, prev->data_size, dropout_rate);
	else
		utils::scaleGpuValue(prev->data, prev->data_size, 1 - dropout_rate);
}

}
