/*
 * input.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "input.cuh"

namespace layer {

Input::Input(int n, int c, int h, int w) : Layer() {
	prev = NULL;

	batch = n;
	callCudnn(hipdnnCreateTensorDescriptor(&t_data));
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW,	HIPDNN_DATA_FLOAT,
			n, c, h, w));
	data_size = n * c * h * w;
	callCuda(hipMalloc(&data, sizeof(float) * data_size));
}

Input::~Input() {
	callCudnn(hipdnnDestroyTensorDescriptor(&t_data));
	callCuda(hipFree(data));
}

void Input::forward() {
	// nothing
}

void Input::backward() {
	// nothing
}

void Input::update(float alpha) {
	// nothing
}

}
