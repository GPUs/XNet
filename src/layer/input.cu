/*
 * input.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "input.cuh"

namespace layer {

Input::Input(int n, int c, int h, int w, float* _data) : Layer() {
	prev = NULL;

	batch = n;
	callCudnn(hipdnnCreateTensorDescriptor(&t_data));
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW,	HIPDNN_DATA_FLOAT,
			n, c, h, w));
	data_size = n * c * h * w;
	data = _data;
	//callCuda(hipMalloc(&data, sizeof(float) * data_size));
	//callCuda(hipMemcpy(data, _data, data_size, hipMemcpyHostToDevice));
}

Input::~Input() {
	callCudnn(hipdnnDestroyTensorDescriptor(t_data));
	//callCuda(hipFree(data));
	data = NULL;
}

void Input::forward() {
	// nothing
}

void Input::backward() {
	// nothing
}

void Input::update(float alpha) {
	// nothing
}

}
