/*
 * input.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "input.cuh"

namespace layer {

Input::Input(int n, int c, int h, int w, float* _data) : Layer() {
	prev = NULL;

	batch = n;
	callCudnn(hipdnnCreateTensorDescriptor(&t_data));
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW,	HIPDNN_DATA_FLOAT,
			n, c, h, w));
	data_size = n * c * h * w;
	data = _data;
}

Input::~Input() {
	callCudnn(hipdnnDestroyTensorDescriptor(t_data));
	data = NULL;
}

void Input::forward() {
	// nothing
}

void Input::backward() {
	// nothing
}

void Input::update(float alpha) {
	// nothing
}

}
