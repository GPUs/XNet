#include "hip/hip_runtime.h"
/*
 * output.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "output.cuh"

using namespace global;

namespace layer {

__global__ void softmaxLoss(const float *label, int label_dim, int batch, float *diff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batch)
		return;

	const int label_value = static_cast<int>(label[idx]);

	diff[idx * label_dim + label_value] -= 1.0f;
}

Output::Output(Layer* _prev, float* _label, int _label_dim, int _batch) : Layer() {
	prev = _prev;
	prev->next = this;

	batch = _batch;
	label_dim = _label_dim;
	callCudnn(hipdnnCreateTensorDescriptor(&t_data));
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			batch, 1, 1, 1));
	data_size = batch;
	callCuda(hipMalloc(&data, sizeof(float) * data_size));
	//callCuda(hipMalloc(&label, sizeof(float) * data_size));
	//callCuda(hipMemcpy(label, _label, batch, hipMemcpyHostToDevice));
	label = _label;

	callCuda(hipMalloc(&diff, sizeof(float) * prev->data_size));
}

Output::~Output() {
	callCudnn(hipdnnDestroyTensorDescriptor(t_data));
	callCuda(hipFree(data));
	//callCuda(hipFree(label));
	label = NULL;
	callCuda(hipFree(diff));
}

void Output::forward() {
	// nothing
}

void Output::backward() {
	//std::cout << "hehre\n";
	utils::printGpuMatrix(label, batch, 1, batch, 2);
	callCuda(hipMemcpy(diff, prev->data, sizeof(float) * prev->data_size,
			hipMemcpyDeviceToDevice));
	softmaxLoss<<< (batch + 127) / 128, 128>>> (label, label_dim, batch, diff);
	utils::printGpuMatrix(diff, batch * 10, 10, batch, 2);
}

void Output::update(float alpha) {
	// nothing
}

}
