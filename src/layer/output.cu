/*
 * output.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "output.cuh"

using namespace global;

namespace layer {

Output::Output(Layer* _prev, float* _label, int n) : Layer() {
	prev = _prev;
	prev->next = this;

	batch = n;
	callCudnn(hipdnnCreateTensorDescriptor(&t_data));
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			n, 1, 1, 1));
	callCuda(hipMalloc(&data, sizeof(float) * n));
	callCuda(hipMalloc(&label, sizeof(float) * n));
	callCuda(hipMemcpy(label, _label, n, hipMemcpyHostToDevice));
}

Output::~Output() {
	callCudnn(hipdnnDestroyTensorDescriptor(t_data));
	callCuda(hipFree(data));
	callCuda(hipFree(label));
}

void Output::forward() {
	// nothing
}

void Output::backward() {

}

void Output::update() {
	// nothing
}

}
