#include "hip/hip_runtime.h"
/*
 * output.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "output.h"

using namespace global;

namespace layer {

__global__ void softmaxLoss(const float *label, int label_dim, int batch, float *diff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batch)
		return;

	const int label_value = static_cast<int>(label[idx]);

	diff[idx * label_dim + label_value] -= 1.0f;
}

__global__ void predict(const float *softmax, int label_dim, int batch, float *data)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batch)
		return;

	int label_value = 0;
	float max = -1;
	for (int i = 0; i < label_dim; i++) {
		if (softmax[idx * label_dim + i] > max) {
			max = softmax[idx * label_dim + i];
			label_value = i;
		}
	}

	data[idx] = (float)label_value;
}

Output::Output(Layer* _prev, float* _label, int _label_dim, int _batch) : Layer() {
	prev = _prev;
	prev->next = this;

	batch = _batch;
	label_dim = _label_dim;
	callCudnn(hipdnnCreateTensorDescriptor(&t_data));
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			batch, 1, 1, 1));
	data_size = batch;
	callCuda(hipMalloc(&data, sizeof(float) * data_size));
	label = _label;

	callCuda(hipMalloc(&diff, sizeof(float) * prev->data_size));

	param_size = 0;
	param_bias_size = 0;
}

Output::~Output() {
	callCudnn(hipdnnDestroyTensorDescriptor(t_data));
	callCuda(hipFree(data));
	label = NULL;
	callCuda(hipFree(diff));
}

void Output::forward(bool train) {
	predict<<< (batch + 127) / 128, 128>>> (prev->data, label_dim, batch, data);
}

void Output::backward() {
	callCuda(hipMemcpy(diff, prev->data, sizeof(float) * prev->data_size,
			hipMemcpyDeviceToDevice));
	softmaxLoss<<< (batch + 127) / 128, 128>>> (label, label_dim, batch, diff);
}

void Output::update() {
	// nothing
}

}
