/*
 * relu.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "relu.cuh"

using namespace global;

namespace layer {

ReLU::ReLU(Layer* _prev, int _output_size, float alpha) :
		Neuron(_prev, _output_size, alpha) {}

ReLU::~ReLU() {}

void ReLU::forward_activation() {
	float a = 1;
	float b = 0;
	callCudnn(hipdnnActivationForward(cudnnHandle, HIPDNN_ACTIVATION_RELU, &a,
			t_data, tmp_data, &b, t_data, data));
}

void ReLU::backward_activation() {
	float a = 1;
	float b = 0;
	callCudnn(hipdnnActivationBackward(cudnnHandle, HIPDNN_ACTIVATION_RELU, &a,
			t_data, data, t_data, next->diff,
			t_data, tmp_data, &b, t_data, tmp_diff));
}

}
