/*
 * pooling.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "pooling.cuh"

using namespace global;

namespace layer {

Pooling::Pooling(Layer* _prev, int size, int stride) {
	prev = _prev;
	prev->next = this;

	callCudnn(hipdnnCreatePoolingDescriptor(&descriptor));
	callCudnn(hipdnnSetPooling2dDescriptor(descriptor, HIPDNN_POOLING_MAX,
			size, size,	0, 0, stride, stride));

	int _n, _c, _h, _w, _tmp;
	hipdnnDataType_t _t;
	callCudnn(hipdnnGetTensor4dDescriptor(prev->t_data, &_t, &_n, &_c, &_h, &_w, &_tmp,
			&_tmp, &_tmp, &_tmp));
	batch = _n;
	callCudnn(hipdnnCreateTensorDescriptor(&t_data));
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
			_n, _c, _h / stride, _w / stride));
	data_size = _n * _c * (_h / stride) * (_w / stride);
	callCuda(hipMalloc(&data, sizeof(float) * data_size));
	callCuda(hipMalloc(&diff, sizeof(float) * prev->data_size));
}

Pooling::~Pooling() {
	callCudnn(hipdnnDestroyPoolingDescriptor(descriptor));
	callCudnn(hipdnnDestroyTensorDescriptor(t_data));
	callCuda(hipFree(data));
	callCuda(hipFree(diff));
}

void Pooling::forward() {
	float a = 1;
	float b = 0;
	callCudnn(hipdnnPoolingForward(cudnnHandle, descriptor, &a, prev->t_data,
			prev->data, &b, t_data, data));
}

void Pooling::backward() {
	float a = 1;
	float b = 0;
	callCudnn(hipdnnPoolingBackward(cudnnHandle, descriptor, &a,
			t_data, data, t_data, next->diff,
			prev->t_data, prev->data, &b, prev->t_data, diff));
}

void Pooling::update(float alpha) {

}

}
