/*
 * convolution.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "convolution.cuh"

using namespace global;

namespace layer {

Convolution::Convolution(Layer* _prev, int n ,int c, int h, int w, int kernel) :
		Layer() {
	prev = _prev;
	prev->next = this;

	batch = n;

	callCudnn(hipdnnCreateConvolutionDescriptor(&descriptor));
	callCudnn(hipdnnSetConvolution2dDescriptor(descriptor, 0, 0,	1, 1, 1, 1,
			HIPDNN_CROSS_CORRELATION));

	int _n, _c, _h, _w, _tmp;
	hipdnnDataType_t _t;
	callCudnn(hipdnnGetTensor4dDescriptor(prev->t_data, &_t, &_n, &_c, &_h, &_w, &_tmp,
			&_tmp, &_tmp, &_tmp));
	callCudnn(hipdnnCreateFilterDescriptor(&filter));
	callCudnn(hipdnnSetFilter4dDescriptor(filter, HIPDNN_DATA_FLOAT,
			c, _c, kernel, kernel));
	int param_size =  _c * c * kernel * kernel;
	callCuda(hipMalloc(&param, sizeof(float) * param_size));
	callCuda(hipMalloc(&gradient, sizeof(float) * param_size));
	utils::setGpuNormalValue(param, param_size);

	callCudnn(hipdnnCreateTensorDescriptor(&t_data));
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW,	HIPDNN_DATA_FLOAT,
			n, c, h, w));
	callCuda(hipMalloc(&data, sizeof(float) * n * c * h * w));
	callCuda(hipMalloc(&diff, sizeof(float) * n * c * h * w));

	callCudnn(hipdnnCreateTensorDescriptor(&t_bias));
	callCudnn(hipdnnSetTensor4dDescriptor(t_bias, HIPDNN_TENSOR_NCHW,	HIPDNN_DATA_FLOAT,
			1, c, 1, 1));
	callCuda(hipMalloc(&param_bias, sizeof(float) * c));
	utils::setGpuNormalValue(param_bias, c);

	callCudnn(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle, prev->t_data, filter,
			descriptor, t_data,	HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));
	callCudnn(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
			prev->t_data, filter, descriptor, t_data, algo, &workspace_size));
	callCuda(hipMalloc(&workspace, workspace_size));
}

Convolution::~Convolution() {
	callCudnn(hipdnnDestroyFilterDescriptor(filter));
	callCudnn(hipdnnDestroyConvolutionDescriptor(descriptor));
	callCudnn(hipdnnDestroyTensorDescriptor(t_data));
	callCudnn(hipdnnDestroyTensorDescriptor(t_bias));
	callCuda(hipFree(data));
	callCuda(hipFree(diff));
	callCuda(hipFree(param));
	callCuda(hipFree(param_bias));
	callCuda(hipFree(gradient));
}

void Convolution::forward() {
	float a = 1;
	float b = 0;
	callCudnn(hipdnnConvolutionForward(cudnnHandle, &a, prev->t_data, prev->data, filter,
			param, descriptor, algo, workspace, workspace_size, &b, t_data, data));
	callCudnn(hipdnnAddTensor(cudnnHandle, CUDNN_ADD_SAME_C, &a, t_bias,	param_bias,
			&a, t_data, data));
}

void Convolution::backward() {

}

void Convolution::update() {

}

}
