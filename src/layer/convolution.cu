/*
 * convolution.cu
 *
 *  Created on: Sep 20, 2015
 *      Author: lyx
 */

#include "convolution.cuh"

using namespace global;

namespace layer {

Convolution::Convolution(Layer& _prev, int n ,int c, int h, int w, int kernel) :
		Layer() {
	prev = _prev;
	_prev.next = this;

	callCudnn(hipdnnSetConvolution2dDescriptor(descriptor, 0, 0,	1, 1, 1, 1,
			HIPDNN_CROSS_CORRELATION));

	int _n, _c, _h, _w, _tmp;
	hipdnnDataType_t _t;
	callCudnn(hipdnnGetTensor4dDescriptor(prev.t_data, &_t, &_n, &_c, &_h, &_w, &_tmp,
			&_tmp, &_tmp, &_tmp));
	callCudnn(hipdnnSetFilter4dDescriptor(filter, HIPDNN_DATA_FLOAT,
			c, _c, kernel, kernel));
	int param_size =  _c * c * kernel * kernel;
	callCuda(hipMalloc(param, sizeof(float) * param_size));
	callCuda(hipMalloc(gradient, sizeof(float) * param_size));
	utils::setGpuNormalValue(param, param_size);

	callCudnn(hipdnnCreateTensorDescriptor(&t_data));
	callCudnn(hipdnnSetTensor4dDescriptor(t_data, HIPDNN_TENSOR_NCHW,	HIPDNN_DATA_FLOAT,
			n, c, h, w));
	callCuda(hipMalloc(data, sizeof(float) * n * c * h * w));
	callCuda(hipMalloc(diff, sizeof(float) * n * c * h * w));

	callCudnn(hipdnnCreateTensorDescriptor(&t_bias));
	callCudnn(hipdnnSetTensor4dDescriptor(t_bias, HIPDNN_TENSOR_NCHW,	HIPDNN_DATA_FLOAT,
			1, c, 1, 1));
	callCuda(hipMalloc(param_bias, sizeof(float) * c));
	utils::setGpuNormalValue(param_bias, c);

	callCudnn(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle, prev.t_data, filter,
			descriptor, t_data,	HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));
	callCudnn(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
			prev.t_data, filter, descriptor, t_data, algo, &workspace_size));
}

Convolution::~Convolution() {
	callCudnn(hipdnnDestroyFilterDescriptor(filter));
	callCudnn(hipdnnDestroyConvolutionDescriptor(descriptor));
	callCudnn(hipdnnDestroyTensorDescriptor(t_data));
	callCudnn(hipdnnDestroyTensorDescriptor(t_bias));
	callCuda(hipFree(data));
	callCuda(hipFree(diff));
	callCuda(hipFree(param));
	callCuda(hipFree(param_bias));
	callCuda(hipFree(gradient));
}

void Convolution::forward() {

}

void Convolution::backward() {

}

void Convolution::update() {

}

}
